
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <thread>

#include <unistd.h>

#define cuda_try(call)                                                                \
  do {                                                                                \
    hipError_t err = static_cast<hipError_t>(call);                                 \
    if (err != hipSuccess) {                                                         \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorName(err)); \
      std::terminate();                                                               \
    }                                                                                 \
  } while (0)

__global__ void transferSharedMem(uint32_t *d_a, size_t num_elements) {
  // Calculate global thread ID
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  __shared__ uint32_t shared_mem[1024];

  if (tid < num_elements) {
    shared_mem[threadIdx.x] = d_a[tid];
    //d_a[tid] = 1;
  }
}

__global__ void extractSharedMem(uint32_t *d_a, size_t num_elements) {
  // Calculate global thread ID
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  __shared__ uint32_t shared_mem[1024];

  if (tid < num_elements) {
    d_a[tid] = shared_mem[threadIdx.x];
  }
  
}

hipDevice_t get_cuda_device(const int device_id, int& sms_count) {
  hipDevice_t device;
  int device_count = 0;

  cuda_try(hipInit(0));  // Flag parameter must be zero
  cuda_try(hipGetDeviceCount(&device_count));

  if (device_count == 0) {
    std::cout << "No CUDA capable device found." << std::endl;
    std::terminate();
  }

  cuda_try(hipDeviceGet(&device, device_id));

  hipDeviceProp_t device_prop;
  hipGetDeviceProperties(&device_prop, device_id);

  sms_count = device_prop.multiProcessorCount;

  std::cout << "Device[" << device_id << "]: " << device_prop.name << '\n';
  std::cout << "SMs count: " << sms_count << '\n';
  std::cout << "Total amount of global memory: " << device_prop.totalGlobalMem << ", " << device_prop.totalGlobalMem / 1048576.0f << " MBytes \n";

  return device;
}

// a 1024-bit random sequence
unsigned int uniq_key[32] = {
        0x63636363U, 0x7c7c7c7cU, 0x77777777U, 0x7b7b7b7bU,
        0xf2f2f2f2U, 0x6b6b6b6bU, 0x6f6f6f6fU, 0xc5c5c5c5U,
        0x30303030U, 0x01010101U, 0x67676767U, 0x2b2b2b2bU,
        0xfefefefeU, 0xd7d7d7d7U, 0xababababU, 0x76767676U,
        0x239c9cbfU, 0x53a4a4f7U, 0xe4727296U, 0x9bc0c05bU,
        0x75b7b7c2U, 0xe1fdfd1cU, 0x3d9393aeU, 0x4c26266aU,
        0x6c36365aU, 0x7e3f3f41U, 0xf5f7f702U, 0x83cccc4fU,
        0x6834345cU, 0x51a5a5f4U, 0xd1e5e534U, 0xf9f1f108U
};

int main() {
  int device_id = 0;
  int sms_count = 0;
  hipDevice_t dev = get_cuda_device(device_id, sms_count);

  size_t total_memsize, available_memsize;

  hipMemGetInfo(&available_memsize, &total_memsize);

  size_t memsize = available_memsize - 65273856;
  std::cout << "Allocating: " << memsize << ", " << memsize / 1048576.0f << " MBytes \n";

  uint32_t *a;
  a = (uint32_t*)malloc(memsize);

  size_t num_elements = memsize / 4;
  std::cout << "Number of elements in a: " << num_elements << "\n";

  for (size_t i = 0; i < memsize/4; i += 32) {
		for (size_t j = 0; j < 32; ++j) {
			a[i+j] = uniq_key[j];
		}
	} 

  uint32_t *d_a;
  cuda_try(hipMalloc(&d_a, memsize));
  std::cout << "Transferring data to GPU\n";
  cuda_try(hipMemcpy(d_a, a, memsize, hipMemcpyHostToDevice)); // fill GPU memory with a predefined value
  //cuda_try(cudaFree(d_a)); // deallocate it!!

  int num_threads = 1024;
  size_t num_blocks = num_elements / num_threads;

  std::cout << "Launching GPU kernel (transferSharedMem)\n";
  transferSharedMem<<<num_blocks, num_threads, num_threads * sizeof(uint32_t)>>>(d_a, num_elements);

  hipDeviceSynchronize();

  /*cudaFree(d_a);

  cuda_try(cudaMalloc(&d_a, memsize));

  for (size_t i = 0; i < num_elements; i++) {
		a[i] = 0;
	} 
  cuda_try(cudaMemcpy(d_a, a, memsize, cudaMemcpyHostToDevice));

  std::cout << "Launching GPU kernel (extractSharedMem)\n";
  extractSharedMem<<<num_blocks, num_threads, num_threads * sizeof(uint32_t)>>>(d_a, num_elements);
  cudaDeviceSynchronize();

  cuda_try(cudaMemcpy(a, d_a, memsize, cudaMemcpyDeviceToHost));

  size_t num_uncleared = 0;
  size_t num_zero = 0;
  size_t num_others = 0;

  std::cout << "Checking outputs\n";

  for (size_t i = 0; i < memsize/4; i+= 32) {
    for (size_t j = 0; j < 32; ++j) {
			if(a[i+j] == uniq_key[j]) {
        num_uncleared++;
      }
      else if(a[i+j] == 0) {
        num_zero++;
      }
      else {
        num_others++;
      }
		}
  }

    std::cout << "Number of uncleared memory: " << num_uncleared << "\n";
  std::cout << "Number of zeros: " << num_zero << "\n";
  std::cout << "Number of others: " << num_others << "\n";*/


  //cuda_try(cudaMemcpy(a, d_a, memsize, cudaMemcpyDeviceToHost));
  //std::cout << "Finished copying data\n";
}