
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <thread>

#include <unistd.h>

#define cuda_try(call)                                                                \
  do {                                                                                \
    hipError_t err = static_cast<hipError_t>(call);                                 \
    if (err != hipSuccess) {                                                         \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorName(err)); \
      std::terminate();                                                               \
    }                                                                                 \
  } while (0)

hipDevice_t get_cuda_device(const int device_id, int& sms_count) {
  hipDevice_t device;
  int device_count = 0;

  cuda_try(hipInit(0));  // Flag parameter must be zero
  cuda_try(hipGetDeviceCount(&device_count));

  if (device_count == 0) {
    std::cout << "No CUDA capable device found." << std::endl;
    std::terminate();
  }

  cuda_try(hipDeviceGet(&device, device_id));

  hipDeviceProp_t device_prop;
  hipGetDeviceProperties(&device_prop, device_id);

  sms_count = device_prop.multiProcessorCount;

  std::cout << "Device[" << device_id << "]: " << device_prop.name << '\n';
  std::cout << "SMs count: " << sms_count << '\n';
  std::cout << "Total amount of global memory: " << device_prop.totalGlobalMem << ", " << device_prop.totalGlobalMem / 1048576.0f << " MBytes \n";

  return device;
}

// a 1024-bit random sequence
unsigned int uniq_key[32] = {
        0x63636363U, 0x7c7c7c7cU, 0x77777777U, 0x7b7b7b7bU,
        0xf2f2f2f2U, 0x6b6b6b6bU, 0x6f6f6f6fU, 0xc5c5c5c5U,
        0x30303030U, 0x01010101U, 0x67676767U, 0x2b2b2b2bU,
        0xfefefefeU, 0xd7d7d7d7U, 0xababababU, 0x76767676U,
        0x239c9cbfU, 0x53a4a4f7U, 0xe4727296U, 0x9bc0c05bU,
        0x75b7b7c2U, 0xe1fdfd1cU, 0x3d9393aeU, 0x4c26266aU,
        0x6c36365aU, 0x7e3f3f41U, 0xf5f7f702U, 0x83cccc4fU,
        0x6834345cU, 0x51a5a5f4U, 0xd1e5e534U, 0xf9f1f108U
};

int main() {
  int device_id = 0;
  int sms_count = 0;
  hipDevice_t dev = get_cuda_device(device_id, sms_count);

  size_t total_memsize, available_memsize;

  hipMemGetInfo(&available_memsize, &total_memsize);
  size_t attacker_base_usage = total_memsize - available_memsize;
  std::cout << "Attacker base memory usage: " << attacker_base_usage << ", " << attacker_base_usage / 1048576.0f << " MBytes \n";

  size_t memsize = available_memsize - 65273856;

  char *a;
  a = (char*)malloc(memsize);

  std::cout << "Number of elements in a: " << memsize / 8 << "\n";

  for (size_t i = 0; i < memsize/8; i += 64) {
		for (size_t j = 0; j < 64; ++j) {
			a[i+j] = 'A';
		}
	} 

  unsigned int *d_a;
  cuda_try(hipMalloc(&d_a, memsize));
  cuda_try(hipMemcpy(d_a, a, memsize, hipMemcpyHostToDevice)); // fill GPU memory with a predefined value
  //cuda_try(cudaFree(d_a)); // deallocate it!!
}